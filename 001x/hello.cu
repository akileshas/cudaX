
#include <hip/hip_runtime.h>
#include <stdio.h>

// create a kernel function
__global__ void ping() {
    printf("ping from [block: %d](thread: %d)\n", blockIdx.x, threadIdx.x);
}

int main() {
    // <<< numBlocks, numThreadsPerBlock >>>
    ping<<<2,5>>>();

    // used to join and synchronize threads
    hipDeviceSynchronize();
    return 0;
}
